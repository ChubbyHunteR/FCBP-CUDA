#include "hip/hip_runtime.h"
#include "PredictorNECUDA.h"
#include "../config.h"

namespace {

	__device__ byte predict(byte *iData, unsigned w, unsigned h) {
		unsigned absolutePosition = threadIdx.x + blockIdx.x * THREADS;
		unsigned x = absolutePosition % w + 1;
		unsigned y = absolutePosition / w - 1;
		if(x < w && y < h){
			return iData[y * w + x];
		}
		return 0;
	}

	__global__ void predict(void *diData, void *dPredicted, unsigned w, unsigned h) {
		unsigned absolutePosition = threadIdx.x + blockIdx.x * THREADS;
		if(absolutePosition >= w*h){
			return;
		}

		byte* iData = (byte*) diData;
		byte* predicted = (byte*) dPredicted;

		predicted[absolutePosition] = predict(iData, w, h);
	}
}

void PredictorNE::cudaPredictAll(void *diData, void *dPredicted, unsigned w, unsigned h){
	unsigned size = w * h;
	::predict<<<size/THREADS + 1, THREADS>>>(diData, dPredicted, w, h);
}
