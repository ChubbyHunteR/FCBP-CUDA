#include "hip/hip_runtime.h"
#include "PredictorW.h"
#include "../config.h"

namespace {

	__device__ byte predict(byte *iData, unsigned w, unsigned h) {
		unsigned absolutePosition = threadIdx.x + blockIdx.x * THREADS;
		unsigned x = absolutePosition % w - 1;
		unsigned y = absolutePosition / w;
		if(x < w && y < h){
			return iData[y * w + x];
		}
		return 0;
	}

	__global__ void predict(void *diData, void *dPredicted, unsigned w, unsigned h) {
		unsigned absolutePosition = threadIdx.x + blockIdx.x * THREADS;
		if(absolutePosition >= w*h){
			return;
		}

		byte* iData = (byte*) diData;
		byte* predicted = (byte*) dPredicted;

		predicted[absolutePosition] = predict(iData, w, h);
	}
}

void PredictorW::cudaPredictAll(void *diData, void *dPredicted, unsigned w, unsigned h){
	unsigned size = w * h;
	::predict<<<size/THREADS + 1, THREADS>>>(diData, dPredicted, w, h);
}
