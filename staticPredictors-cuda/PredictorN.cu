#include "hip/hip_runtime.h"
#include "PredictorN.h"
#include "../config.h"

namespace {

	__device__ byte predict(byte *iData, unsigned x, unsigned y, unsigned w, unsigned h) {
		--y;
		if(x < w && y < h){
			return iData[y * w + x];
		}
		return 0;
	}

	__global__ void predict(void *diData, void *dPredicted, unsigned w, unsigned h) {
		unsigned absolutePosition = threadIdx.x + blockIdx.x * THREADS;
		unsigned x = absolutePosition % w;
		unsigned y = absolutePosition / w;
		if(x < w && y < h){
			byte* iData = (byte*) diData;
			byte* predicted = (byte*) dPredicted;
			predicted[absolutePosition] = predict(iData, x, y, w, h);
		}
	}
}

void PredictorN::cudaPredictAll(void *diData, void *dPredicted, unsigned w, unsigned h){
	unsigned size = w * h;
	::predict<<<size/THREADS + 1, THREADS>>>(diData, dPredicted, w, h);
}
