#include "hip/hip_runtime.h"
#include "PGMCBPCCUDA.h"
#include <iostream>

#define SWAP(a, b) (((a) ^= (b)), ((b) ^= (a)), ((a) ^= (b)))

namespace {
	__device__ void insert(unsigned dist, unsigned x, unsigned y, unsigned similarPixelsX[], unsigned similarPixelsY[], unsigned similarPixelsDistance[], unsigned* numOfSimilarPixels){
		unsigned tmpDist = dist;
		unsigned tmpPixelX = x;
		unsigned tmpPixelY = y;

		for(int i = 0; i < *numOfSimilarPixels; ++i){
			if(tmpDist > similarPixelsDistance[i]){
				SWAP(tmpDist, similarPixelsDistance[i]);
				SWAP(tmpPixelX, similarPixelsX[i]);
				SWAP(tmpPixelY, similarPixelsY[i]);
			}
		}

		if(*numOfSimilarPixels < M){
			similarPixelsDistance[*numOfSimilarPixels] = tmpDist;
			similarPixelsX[*numOfSimilarPixels] = tmpPixelX;
			similarPixelsY[*numOfSimilarPixels] = tmpPixelY;
			++*numOfSimilarPixels;
		}
	}

	__device__ unsigned distance(	byte* iData,
									unsigned anchorX,
									unsigned anchorY,
									unsigned x,
									unsigned y,
									PixelOffset* vectorOffset,
									unsigned w,
									unsigned h)
	{
		unsigned x1, x2, y1, y2, sum = 0;
		int pix1, pix2;
		for(int i = 0; i < D; ++i){
			x1 = anchorX + vectorOffset[i].x;
			y1 = anchorY + vectorOffset[i].y;
			x2 = x + vectorOffset[i].x;
			y2 = y + vectorOffset[i].y;
			pix1 = pix2 = 0;
			if(x1 < w && y1 < h){
				pix1 = iData[x1 + y1 * w];
			}
			if(x2 < w && y2 < h){
				pix2 = iData[x2 + y2 * w];
			}
			sum += (pix1-pix2) * (pix1-pix2);
		}
		return sum;
	}

	__device__ byte predict(byte* iData,
							byte** predicted,
							unsigned numOfPredictors,
							PixelOffset* radiusOffset,
							PixelOffset* vectorOffset,
							unsigned w,
							unsigned h)
	{
		unsigned absolutePosition = threadIdx.x + blockIdx.x * THREADS;
		unsigned anchorX = absolutePosition % w;
		unsigned anchorY = absolutePosition / w;
		unsigned x, y, dist, numOfSimilarPixels = 0;
		unsigned similarPixelsDistance[M];
		unsigned similarPixelsX[M];
		unsigned similarPixelsY[M];
		float* penalties = new float[numOfPredictors];

		for(int i = 0; i < R_A; ++i){
			x = anchorX + radiusOffset[i].x;
			y = anchorY + radiusOffset[i].y;
			if(x < w && y < h){
				dist = distance(iData, anchorX, anchorY, x, y, vectorOffset, w, h);
				insert(dist, x, y, similarPixelsX, similarPixelsY, similarPixelsDistance, &numOfSimilarPixels);
			}
		}

		for(int i = 0; i < numOfPredictors; ++i){
			unsigned sum = 0;
			for(int j = 0; j < numOfSimilarPixels; ++j){
				int prediction = predicted[i][similarPixelsX[j] + similarPixelsY[j] * w ];
				int pixel = iData[ similarPixelsX[j] + similarPixelsY[j] * w ];
				sum += (prediction - pixel) * (prediction - pixel);
			}
			if(numOfSimilarPixels == 0){
				penalties[i] = 0;
			}else{
				penalties[i] = (float)sum / numOfSimilarPixels;
			}
		}

		float sum = 0;
		float penaltiesSum = 0;
		for(int i = 0; i < numOfPredictors; ++i){
			sum += predicted[i][absolutePosition] / penalties[i];
			penaltiesSum += 1 / penalties[i];
		}

		delete penalties;
		if(penaltiesSum == 0){
			return 0;
		}else{
			return sum / penaltiesSum;
		}
	}

	__global__ void predict(void* diData,
							void* doData,
							void* deData,
							void** dPredicted,
							unsigned numOfPredictors,
							void* dRadiusOffset,
							void* dVectorOffset,
							ImageWHSize imageMeta)
	{
		unsigned absolutePosition = threadIdx.x + blockIdx.x * THREADS;
		if(absolutePosition >= imageMeta.size){
			return;
		}

		byte* iData = (byte*) diData;
		byte* oData = (byte*) doData;
		short* eData = (short*) deData;
		byte** predicted = (byte**) dPredicted;

		PixelOffset* radiusOffset = (PixelOffset*) dRadiusOffset;
		PixelOffset* vectorOffset = (PixelOffset*) dVectorOffset;

		short prediction = predict(iData, predicted, numOfPredictors, radiusOffset, vectorOffset, imageMeta.w, imageMeta.h);
		oData[absolutePosition] = prediction;
		eData[absolutePosition] = iData[absolutePosition] - prediction;
	}
}

PGMCBPCCUDA::PGMCBPCCUDA(	vector<PGMImage>& inputImages,
							vector<PGMImage>& outputImages,
							vector<PGMImage>& errorImages,
							vector<Predictor*>& predictors
						):
		inputImages(inputImages),
		outputImages(outputImages),
		errorImages(errorImages),
		predictors(predictors)
{
	for(auto& inputImage : inputImages){
		imagesMeta.emplace_back(inputImage.getWidth(), inputImage.getHeight(), inputImage.getSize());
		iData.push_back(inputImage.getBuffer());
	}
	for(auto& outputImage : outputImages){
		oData.push_back(outputImage.getBuffer());
	}
	for(auto& imageMeta : imagesMeta){
		eData.push_back(new short[imageMeta.size]);
	}

	for(int i = 0; i < R_A; ++i){
		radiusOffset[i].x = i % (2*R + 1) - R;
		radiusOffset[i].y = i / (2*R + 1) - R;
	}
	for(int i = 0; i < D; ++i){
		vectorOffset[i].x = i % (D_R + 2) - D_R;
		vectorOffset[i].y = i / (D_R + 2) - D_R;
	}

	for(int i = 0; i < iData.size(); ++i){
		cout<<"Static prediction "<<i+1<<"/"<<iData.size()<<endl;
		diData.push_back(NULL);
		doData.push_back(NULL);
		deData.push_back(NULL);
		dPredicted.push_back(NULL);

		CUDA_CHECK_RETURN(hipMalloc(&diData[i], sizeof(byte) * imagesMeta[i].size));
		CUDA_CHECK_RETURN(hipMalloc(&doData[i], sizeof(byte) * imagesMeta[i].size));
		CUDA_CHECK_RETURN(hipMalloc(&deData[i], sizeof(short) * imagesMeta[i].size));
		CUDA_CHECK_RETURN(hipMemcpy(diData[i], iData[i], sizeof(byte) * imagesMeta[i].size, hipMemcpyHostToDevice));

		CUDA_CHECK_RETURN(hipMalloc(&dPredicted[i], sizeof(void*) * predictors.size()));
		vector<void*> hPredicted(predictors.size());
		for(int j = 0; j < predictors.size(); ++j){
			CUDA_CHECK_RETURN(hipMalloc(&hPredicted[j], sizeof(byte) * imagesMeta[i].size));
			predictors[j]->predict(diData[i], hPredicted[j], imagesMeta[i].w, imagesMeta[i].h);
		}
		CUDA_CHECK_RETURN(hipMemcpy(dPredicted[i], hPredicted.data(), sizeof(void*) * predictors.size(), hipMemcpyHostToDevice));
		cout<<"DONE"<<endl;
	}

	CUDA_CHECK_RETURN(hipMalloc(&dRadiusOffset, sizeof(radiusOffset)));
	CUDA_CHECK_RETURN(hipMemcpy(dRadiusOffset, radiusOffset, sizeof(radiusOffset), hipMemcpyHostToDevice));

	CUDA_CHECK_RETURN(hipMalloc(&dVectorOffset, sizeof(vectorOffset)));
	CUDA_CHECK_RETURN(hipMemcpy(dVectorOffset, vectorOffset, sizeof(vectorOffset), hipMemcpyHostToDevice));
}

PGMCBPCCUDA::~PGMCBPCCUDA(){
	CUDA_CHECK_RETURN(hipDeviceReset());
	for(auto p : eData){
		delete[] p;
	}
}

void PGMCBPCCUDA::predict(){
	for(int i = 0; i < inputImages.size(); ++i){
		cout<<"Prediction "<<i+1<<"/"<<inputImages.size()<<endl;
		::predict<<<imagesMeta[i].size/THREADS + 1, THREADS>>>(		diData[i],
																	doData[i],
																	deData[i],
																	dPredicted[i],
																	predictors.size(),
																	dRadiusOffset,
																	dVectorOffset,
																	imagesMeta[i]
																);
		CUDA_CHECK_RETURN(hipMemcpy(oData[i], doData[i], sizeof(byte) * imagesMeta[i].size, hipMemcpyDeviceToHost));
		CUDA_CHECK_RETURN(hipMemcpy(eData[i], deData[i], sizeof(short) * imagesMeta[i].size, hipMemcpyDeviceToHost));

		for(unsigned k = 0; k < imagesMeta[i].h; ++k){
			for(unsigned j = 0; j < imagesMeta[i].w; ++j){
				short error = eData[i][j + k*imagesMeta[i].w];
				if(error < 0){
					error = -error;
				}
				errorImages[i].writePixel(j, k, error);
			}
		}
		cout<<"DONE"<<endl;
	}
}

bool PGMCBPCCUDA::getStaticPrediction(unsigned predictorIndex){
	if(predictorIndex >= predictors.size()){
		return false;
	}

	for(int i = 0; i < inputImages.size(); ++i){
		void** hPredicted = new void*[predictors.size()];
		CUDA_CHECK_RETURN(hipMemcpy(hPredicted, dPredicted[i], sizeof(void*) * predictors.size(), hipMemcpyDeviceToHost));
		CUDA_CHECK_RETURN(hipMemcpy(oData[i], hPredicted[predictorIndex], sizeof(byte) * imagesMeta[i].size, hipMemcpyDeviceToHost));
		delete[] hPredicted;
	}

	return true;
}
