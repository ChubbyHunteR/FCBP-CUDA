#include "hip/hip_runtime.h"
#include <iostream>
#include "PGMCBPDCUDA.h"
#include "decoderPredictors/predictors.h"

#define SWAP(a, b) (((a) ^= (b)), ((b) ^= (a)), ((a) ^= (b)))

struct PixelDistance{
	unsigned x, y, distance;
	__device__ PixelDistance(unsigned x, unsigned y, unsigned distance) : x(x), y(y), distance(distance){}
	__device__ PixelDistance() : x(0), y(0), distance(0){}
};

namespace {
	__device__ void insert(PixelDistance pixelDist, PixelDistance similarPixels[M], unsigned* numOfSimilarPixels){
		for(int i = 0; i < *numOfSimilarPixels; ++i){
			if(pixelDist.distance < similarPixels[i].distance){
				SWAP(pixelDist.distance, similarPixels[i].distance);
				SWAP(pixelDist.x, similarPixels[i].x);
				SWAP(pixelDist.y, similarPixels[i].y);
			}
		}

		if(*numOfSimilarPixels < M){
			similarPixels[*numOfSimilarPixels] = pixelDist;
			++*numOfSimilarPixels;
		}
	}

	__device__ unsigned distance(	byte* iData,
									unsigned anchorX,
									unsigned anchorY,
									unsigned x,
									unsigned y,
									PixelOffset* vectorOffset,
									unsigned w,
									unsigned h)
	{
		unsigned x1, x2, y1, y2;
		int sum = 0, pix1, pix2;
		for(int i = 0; i < D; ++i){
			x1 = anchorX + vectorOffset[i].x;
			y1 = anchorY + vectorOffset[i].y;
			x2 = x + vectorOffset[i].x;
			y2 = y + vectorOffset[i].y;
			pix1 = pix2 = 0;
			if(x1 < w && y1 < h){
				pix1 = iData[x1 + y1 * w];
			}
			if(x2 < w && y2 < h){
				pix2 = iData[x2 + y2 * w];
			}
			sum += (pix1-pix2) * (pix1-pix2);
		}
		return sum;
	}

	__device__ byte predict(byte* oData,
							unsigned anchorX,
							unsigned anchorY,
							unsigned w,
							unsigned h,
							PixelOffset* radiusOffset,
							PixelOffset* vectorOffset)
	{
		unsigned numOfSimilarPixels = 0;
		PixelDistance similarPixels[M];
		PixelDistance pixelDist;

		for(int i = 0; i < R_A; ++i){
			pixelDist.x = anchorX + radiusOffset[i].x;
			pixelDist.y = anchorY + radiusOffset[i].y;
			if(pixelDist.x < w && pixelDist.y < h){
				pixelDist.distance = distance(oData, anchorX, anchorY, pixelDist.x, pixelDist.y, vectorOffset, w, h);
				insert(pixelDist, similarPixels, &numOfSimilarPixels);
			}
		}
		if(numOfSimilarPixels == 0){
			return 0;
		}

		float* penalties = new float[numOfPredictors];
		for(int i = 0; i < numOfPredictors; ++i){
			unsigned sum = 0;
			for(int j = 0; j < numOfSimilarPixels; ++j){
				int prediction = predicted[i][similarPixels[j].x + similarPixels[j].y * w ];
				int pixel = iData[ similarPixels[j].x + similarPixels[j].y * w ];
				sum += (prediction - pixel) * (prediction - pixel);
			}
			if(sum == 0){
				delete[] penalties;
				return predicted[i][absolutePosition];
			}
			penalties[i] = (float)sum / numOfSimilarPixels;
		}

		float sum = 0;
		float penaltiesSum = 0;
		for(int i = 0; i < numOfPredictors; ++i){
			sum += predicted[i][absolutePosition] / penalties[i];
			penaltiesSum += 1 / penalties[i];
		}
		delete[] penalties;
		return sum / penaltiesSum;
	}

	__global__ void decode(	void* diData,
							void* doData,
							void* dpData,
							void* dRadiusOffset,
							void* dVectorOffset,
							ImageWHSize imageMeta)
	{
		short* iData = (short*) diData;
		byte* oData = (byte*) doData;
		byte* pData = (byte*) dpData;
		PixelOffset* radiusOffset = (PixelOffset*) dRadiusOffset;
		PixelOffset* vectorOffset = (PixelOffset*) dVectorOffset;

		for(unsigned y = 0; y < imageMeta.h; ++y){
			for(unsigned x = 0; x < imageMeta.w; ++x){
				unsigned pos = x + y * imageMeta.w;
				byte prediction = predict(oData, x, y, imageMeta.w, imageMeta.h, radiusOffset, vectorOffset);
				// TODO errorCorrect()
				pData[pos] = prediction;
				oData[pos] = prediction + iData[pos];
			}
		}
	}

	__global__ void errorCorrect(	void* diData,
									void* dpData,
									void* doData,
									void* deData,
									void* dRadiusOffset,
									void* dVectorOffset,
									ImageWHSize imageMeta)
	{
		unsigned absolutePosition = threadIdx.x + blockIdx.x * THREADS;
		unsigned anchorX = absolutePosition % imageMeta.w;
		unsigned anchorY = absolutePosition / imageMeta.w;
		if(anchorX >= imageMeta.w || anchorY >= imageMeta.h){
			return;
		}
		unsigned numOfSimilarPixels = 0;
		PixelDistance similarPixels[M];
		PixelDistance pixelDist;

		byte* iData = (byte*) diData;
		byte* pData = (byte*) dpData;
		byte* oData = (byte*) doData;
		short* eData = (short*) deData;
		PixelOffset* radiusOffset = (PixelOffset*) dRadiusOffset;
		PixelOffset* vectorOffset = (PixelOffset*) dVectorOffset;

		for(int i = 0; i < R_A; ++i){
			pixelDist.x = anchorX + radiusOffset[i].x;
			pixelDist.y = anchorY + radiusOffset[i].y;
			if(pixelDist.x < imageMeta.w && pixelDist.y < imageMeta.h){
				pixelDist.distance = distance(iData, anchorX, anchorY, pixelDist.x, pixelDist.y, vectorOffset, imageMeta.w, imageMeta.h);
				insert(pixelDist, similarPixels, &numOfSimilarPixels);
			}
		}
		if(numOfSimilarPixels == 0){
			oData[absolutePosition] = pData[absolutePosition];
			eData[absolutePosition] = iData[absolutePosition] - pData[absolutePosition];
		}

		int errorSum = 0;
		for(int i = 0; i < numOfSimilarPixels; ++i){
			errorSum += pData[ similarPixels[i].x + similarPixels[i].y * imageMeta.w ] - iData[ similarPixels[i].x + similarPixels[i].y * imageMeta.w ];
		}
		int prediction = (int)pData[absolutePosition] + errorSum / (int)numOfSimilarPixels;
		if(prediction < 0){
			prediction = 0;
		}else if(prediction > 255){
			prediction = 255;
		}
		oData[absolutePosition] = prediction;
		eData[absolutePosition] = iData[absolutePosition] - prediction;
	}
}

PGMCBPDCUDA::PGMCBPDCUDA(	vector<PGMImageError>& inputImagesError,
							vector<PGMImage>& outputImages,
							vector<PGMImage>& predictionImages,
							vector<Predictor*>& predictors
						):
		inputImagesError(inputImagesError),
		outputImages(outputImages),
		predictionImages(predictionImages),
		predictors(predictors)
{
	for(auto& inputImageError : inputImagesError){
		streams.emplace_back();
		hipStreamCreate(&streams.back());
		imagesMeta.emplace_back(inputImageError.getWidth(), inputImageError.getHeight(), inputImageError.getSize());
		iData.push_back(inputImageError.getBuffer());
	}
	for(auto& outputImage : outputImages){
		oData.push_back(outputImage.getBuffer());
	}
	for(auto& predictionImage : predictionImages){
		pData.push_back(predictionImage.getBuffer());
	}

	for(int i = 0; i < R_A; ++i){
		radiusOffset[i].x = i % (2*R + 1) - R;
		radiusOffset[i].y = i / (2*R + 1) - R;
	}
	for(int i = 0; i < D; ++i){
		vectorOffset[i].x = i % (D_R + 2) - D_R;
		vectorOffset[i].y = i / (D_R + 2) - D_R;
	}

	for(int i = 0; i < iData.size(); ++i){
		cout<<"Memory allocation "<<i+1<<"/"<<iData.size()<<endl;

		diData.push_back(NULL);
		doData.push_back(NULL);
		dpData.push_back(NULL);

		CUDA_CHECK_RETURN(hipMalloc(&diData[i], sizeof(short) * imagesMeta[i].size));
		CUDA_CHECK_RETURN(hipMalloc(&doData[i], sizeof(byte) * imagesMeta[i].size));
		CUDA_CHECK_RETURN(hipMalloc(&dpData[i], sizeof(byte) * imagesMeta[i].size));
		CUDA_CHECK_RETURN(hipMemcpy(diData[i], iData[i], sizeof(short) * imagesMeta[i].size, hipMemcpyHostToDevice));

		cout<<"DONE"<<endl;
	}

	CUDA_CHECK_RETURN(hipMalloc(&dRadiusOffset, sizeof(radiusOffset)));
	CUDA_CHECK_RETURN(hipMemcpy(dRadiusOffset, radiusOffset, sizeof(radiusOffset), hipMemcpyHostToDevice));

	CUDA_CHECK_RETURN(hipMalloc(&dVectorOffset, sizeof(vectorOffset)));
	CUDA_CHECK_RETURN(hipMemcpy(dVectorOffset, vectorOffset, sizeof(vectorOffset), hipMemcpyHostToDevice));
}

PGMCBPDCUDA::~PGMCBPDCUDA(){
	CUDA_CHECK_RETURN(hipDeviceReset());
}

void PGMCBPDCUDA::decode(){
	for(int i = 0; i < streams.size(); ++i){
		::decode<<<1, 1, 0, streams[i]>>>(	diData[i],
											doData[i],
											dpData[i],
											dRadiusOffset,
											dVectorOffset,
											imagesMeta[i]);
		CUDA_CHECK_RETURN(hipStreamDestroy(streams[i]));
	}

	for(int i = 0; i < streams.size(); ++i){
		CUDA_CHECK_RETURN(hipStreamSynchronize(streams[i]));
		CUDA_CHECK_RETURN(hipMemcpy(oData[i], doData[i], sizeof(byte) * imagesMeta[i].size, hipMemcpyDeviceToHost));
		CUDA_CHECK_RETURN(hipMemcpy(pData[i], dpData[i], sizeof(byte) * imagesMeta[i].size, hipMemcpyDeviceToHost));
	}
}
