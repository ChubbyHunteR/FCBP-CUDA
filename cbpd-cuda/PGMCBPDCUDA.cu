#include "hip/hip_runtime.h"
#include <iostream>

#include "PGMCBPDCUDA.h"

#define SWAP(a, b) (((a) ^= (b)), ((b) ^= (a)), ((a) ^= (b)))
#define NUM_PREDICTORS 7

struct PixelDistance{
	unsigned x, y, distance;
	__device__ PixelDistance(unsigned x, unsigned y, unsigned distance) : x(x), y(y), distance(distance){}
	__device__ PixelDistance() : x(0), y(0), distance(0){}
};

namespace {
	__device__ byte predictGN(byte *oData, unsigned x, unsigned y, unsigned w, unsigned h) {
		y -= 2;
		int sum = 0;
		if(x < w && y < h){
			sum -= oData[y * w + x];
		}
		++y;
		if(x < w && y < h){
			sum += 2 * oData[y * w + x];
		}

		if(sum < 0){
			sum = 0;
		}else if(sum > 255){
			sum = 255;
		}
		return sum;
	}

	__device__ byte predictGW(byte *oData, unsigned x, unsigned y, unsigned w, unsigned h) {
		x -= 2;
		int sum = 0;
		if(x < w && y < h){
			sum -= oData[y * w + x];
		}
		++x;
		if(x < w && y < h){
			sum += 2 * oData[y * w + x];
		}

		if(sum < 0){
			sum = 0;
		}else if(sum > 255){
			sum = 255;
		}
		return sum;
	}

	__device__ byte predictN(byte *oData, unsigned x, unsigned y, unsigned w, unsigned h) {
		--y;
		if(x < w && y < h){
			return oData[y * w + x];
		}
		return 0;
	}

	__device__ byte predictNE(byte *oData, unsigned x, unsigned y, unsigned w, unsigned h) {
		++x;
		--y;
		if(x < w && y < h){
			return oData[y * w + x];
		}
		return 0;
	}

	__device__ byte predictNW(byte *oData, unsigned x, unsigned y, unsigned w, unsigned h) {
		--x;
		--y;
		if(x < w && y < h){
			return oData[y * w + x];
		}
		return 0;
	}

	__device__ byte predictPL(byte *oData, unsigned x, unsigned y, unsigned w, unsigned h) {
		--x;
		--y;
		int sum = 0;
		if(x < w && y < h){
			sum -= oData[y * w + x];
		}
		++x;
		if(x < w && y < h){
			sum += oData[y * w + x];
		}
		--x;
		++y;
		if(x < w && y < h){
			sum += oData[y * w + x];
		}

		if(sum < 0){
			sum = 0;
		}else if(sum > 255){
			sum = 255;
		}
		return sum;
	}

	__device__ byte predictW(byte *oData, unsigned x, unsigned y, unsigned w, unsigned h) {
		--x;
		if(x < w && y < h){
			return oData[y * w + x];
		}
		return 0;
	}

	__device__ byte (* predictors[])(byte *oData, unsigned x, unsigned y, unsigned w, unsigned h)
	{
		predictN,
		predictNW,
		predictGW,
		predictW,
		predictNE,
		predictGN,
		predictPL,
		NULL
	};

	__device__ void insert(PixelDistance pixelDist, PixelDistance similarPixels[M], unsigned* numOfSimilarPixels){
		for(int i = 0; i < *numOfSimilarPixels; ++i){
			if(pixelDist.distance < similarPixels[i].distance){
				SWAP(pixelDist.distance, similarPixels[i].distance);
				SWAP(pixelDist.x, similarPixels[i].x);
				SWAP(pixelDist.y, similarPixels[i].y);
			}
		}

		if(*numOfSimilarPixels < M){
			similarPixels[*numOfSimilarPixels] = pixelDist;
			++*numOfSimilarPixels;
		}
	}

	__device__ unsigned distance(	byte* iData,
									unsigned anchorX,
									unsigned anchorY,
									unsigned x,
									unsigned y,
									PixelOffset* vectorOffset,
									unsigned w,
									unsigned h)
	{
		unsigned x1, x2, y1, y2;
		int sum = 0, pix1, pix2;
		for(int i = 0; i < D; ++i){
			x1 = anchorX + vectorOffset[i].x;
			y1 = anchorY + vectorOffset[i].y;
			x2 = x + vectorOffset[i].x;
			y2 = y + vectorOffset[i].y;
			pix1 = pix2 = 0;
			if(x1 < w && y1 < h){
				pix1 = iData[x1 + y1 * w];
			}
			if(x2 < w && y2 < h){
				pix2 = iData[x2 + y2 * w];
			}
			sum += (pix1-pix2) * (pix1-pix2);
		}
		return sum;
	}

	__device__ byte predict(byte* oData,
							short* pData,
							unsigned anchorX,
							unsigned anchorY,
							PixelOffset* radiusOffset,
							PixelOffset* vectorOffset,
							ImageWHSize imageMeta)
	{
		unsigned numOfSimilarPixels = 0;
		PixelDistance similarPixels[M];
		PixelDistance pixelDist;

		for(int i = 0; i < R_A; ++i){
			pixelDist.x = anchorX + radiusOffset[i].x;
			pixelDist.y = anchorY + radiusOffset[i].y;
			if(pixelDist.x < imageMeta.w && pixelDist.y < imageMeta.h){
				pixelDist.distance = distance(oData, anchorX, anchorY, pixelDist.x, pixelDist.y, vectorOffset, imageMeta.w, imageMeta.h);
				insert(pixelDist, similarPixels, &numOfSimilarPixels);
			}
		}
		if(numOfSimilarPixels == 0){
			return 0;
		}

		float penalties[NUM_PREDICTORS];
		for(int i = 0; i < NUM_PREDICTORS; ++i){
			unsigned sum = 0;
			int staticPrediction = -1;
			for(int j = 0; j < numOfSimilarPixels; ++j){
				staticPrediction = pData[i * imageMeta.size + similarPixels[j].x + similarPixels[j].y * imageMeta.w];
				if(staticPrediction == -1){
					pData[i * imageMeta.size + similarPixels[j].x + similarPixels[j].y * imageMeta.w] =
					staticPrediction =
					predictors[i](oData, similarPixels[j].x, similarPixels[j].y, imageMeta.w, imageMeta.h);
				}
				int pixel = oData[ similarPixels[j].x + similarPixels[j].y * imageMeta.w ];
				sum += (staticPrediction - pixel) * (staticPrediction - pixel);
			}
			if(sum == 0){
				return pData[i * imageMeta.size + anchorX + anchorY * imageMeta.w] =
						predictors[i](oData, anchorX, anchorY, imageMeta.w, imageMeta.h);
			}
			penalties[i] = (float)sum / numOfSimilarPixels;
		}

		float sum = 0;
		float penaltiesSum = 0;
		for(int i = 0; i < NUM_PREDICTORS; ++i){
			int prediction = pData[i * imageMeta.size + anchorX + anchorY * imageMeta.w] =
					predictors[i](oData, anchorX, anchorY, imageMeta.w, imageMeta.h);
			sum += prediction / penalties[i];
			penaltiesSum += 1 / penalties[i];
		}
		return sum / penaltiesSum;
	}

	__device__ int errorCorrect(	byte* oData,
									short* pData,
									byte* spData,
									unsigned anchorX,
									unsigned anchorY,
									unsigned w,
									unsigned h,
									void* dRadiusOffset,
									void* dVectorOffset)
	{
		unsigned numOfSimilarPixels = 0;
		PixelDistance similarPixels[M];
		PixelDistance pixelDist;

		PixelOffset* radiusOffset = (PixelOffset*) dRadiusOffset;
		PixelOffset* vectorOffset = (PixelOffset*) dVectorOffset;

		for(int i = 0; i < R_A; ++i){
			pixelDist.x = anchorX + radiusOffset[i].x;
			pixelDist.y = anchorY + radiusOffset[i].y;
			if(pixelDist.x < w && pixelDist.y < h){
				pixelDist.distance = distance(oData, anchorX, anchorY, pixelDist.x, pixelDist.y, vectorOffset, w, h);
				insert(pixelDist, similarPixels, &numOfSimilarPixels);
			}
		}
		if(numOfSimilarPixels == 0){
			return 0;
		}

		int errorSum = 0;
		for(int i = 0; i < numOfSimilarPixels; ++i){
			errorSum += spData[ similarPixels[i].x + similarPixels[i].y * w ] - oData[ similarPixels[i].x + similarPixels[i].y * w ];
		}
		return errorSum / (int)numOfSimilarPixels;
	}

	__global__ void decode(	void* diData,
							void* doData,
							void* dpData,
							void* dspData,
							void* dRadiusOffset,
							void* dVectorOffset,
							ImageWHSize imageMeta)
	{
		short* iData = (short*) diData;
		byte* oData = (byte*) doData;
		short* pData = (short*) dpData;
		byte* spData = (byte*) dspData;
		PixelOffset* radiusOffset = (PixelOffset*) dRadiusOffset;
		PixelOffset* vectorOffset = (PixelOffset*) dVectorOffset;

		for(unsigned y = 0; y < imageMeta.h; ++y){
			for(unsigned x = 0; x < imageMeta.w; ++x){
				unsigned pos = x + y * imageMeta.w;
				int prediction = spData[pos] = predict(oData, pData, x, y, radiusOffset, vectorOffset, imageMeta);
				prediction += errorCorrect(oData, pData, spData, x, y, imageMeta.w, imageMeta.h, radiusOffset, vectorOffset);
				if(prediction < 0){
					prediction = 0;
				}else if(prediction > 255){
					prediction = 255;
				}
				oData[pos] = prediction + iData[pos];
			}
		}
	}
}

PGMCBPDCUDA::PGMCBPDCUDA(	vector<PGMImageError>& inputImagesError,
							vector<PGMImage>& outputImages
						):
		inputImagesError(inputImagesError),
		outputImages(outputImages)
{
	for(auto& inputImageError : inputImagesError){
		streams.emplace_back();
		hipStreamCreate(&streams.back());
		imagesMeta.emplace_back(inputImageError.getWidth(), inputImageError.getHeight(), inputImageError.getSize());
		iData.push_back(inputImageError.getBuffer());
		pData.push_back(new short[NUM_PREDICTORS * inputImageError.getSize()]);
	}
	for(auto& outputImage : outputImages){
		oData.push_back(outputImage.getBuffer());
	}

	for(int i = 0; i < R_A; ++i){
		radiusOffset[i].x = i % (2*R + 1) - R;
		radiusOffset[i].y = i / (2*R + 1) - R;
	}
	for(int i = 0; i < D; ++i){
		vectorOffset[i].x = i % (D_R + 2) - D_R;
		vectorOffset[i].y = i / (D_R + 2) - D_R;
	}

	for(int i = 0; i < iData.size(); ++i){
		cout<<"Memory allocation "<<i+1<<"/"<<iData.size()<<endl;

		diData.push_back(nullptr);
		doData.push_back(nullptr);
		dpData.push_back(nullptr);
		dspData.push_back(nullptr);

		CUDA_CHECK_RETURN(hipMalloc(&diData[i], sizeof(short) * imagesMeta[i].size));
		CUDA_CHECK_RETURN(hipMalloc(&doData[i], sizeof(byte) * imagesMeta[i].size));
		CUDA_CHECK_RETURN(hipMalloc(&dpData[i], NUM_PREDICTORS * sizeof(short) * imagesMeta[i].size));
		CUDA_CHECK_RETURN(hipMalloc(&dspData[i], sizeof(byte) * imagesMeta[i].size));
		CUDA_CHECK_RETURN(hipMemset(dpData[i], -1, NUM_PREDICTORS * sizeof(short) * imagesMeta[i].size));
		CUDA_CHECK_RETURN(hipMemcpy(diData[i], iData[i], sizeof(short) * imagesMeta[i].size, hipMemcpyHostToDevice));

		cout<<"DONE"<<endl;
	}

	CUDA_CHECK_RETURN(hipMalloc(&dRadiusOffset, sizeof(radiusOffset)));
	CUDA_CHECK_RETURN(hipMemcpy(dRadiusOffset, radiusOffset, sizeof(radiusOffset), hipMemcpyHostToDevice));

	CUDA_CHECK_RETURN(hipMalloc(&dVectorOffset, sizeof(vectorOffset)));
	CUDA_CHECK_RETURN(hipMemcpy(dVectorOffset, vectorOffset, sizeof(vectorOffset), hipMemcpyHostToDevice));
}

PGMCBPDCUDA::~PGMCBPDCUDA(){
	for(auto& stream : streams){
		CUDA_CHECK_RETURN(hipStreamDestroy(stream));
	}
	CUDA_CHECK_RETURN(hipDeviceReset());
}

void PGMCBPDCUDA::decode(){
	for(int i = 0; i < streams.size(); ++i){
		::decode<<<1, 1, 0, streams[i]>>>(	diData[i],
											doData[i],
											dpData[i],
											dspData[i],
											dRadiusOffset,
											dVectorOffset,
											imagesMeta[i]);
	}

	for(int i = 0; i < streams.size(); ++i){
		cout << "Waiting for " << i+1 << "/" << streams.size() << endl;
		CUDA_CHECK_RETURN(hipStreamSynchronize(streams[i]));
		cout << "DONE" << endl;
	}

	for(int i = 0; i < streams.size(); ++i){
		cout << "Copying " << i+1 << "/" << streams.size() << endl;
		CUDA_CHECK_RETURN(hipMemcpy(oData[i], doData[i], sizeof(byte) * imagesMeta[i].size, hipMemcpyDeviceToHost));
		CUDA_CHECK_RETURN(hipMemcpy(pData[i], dpData[i], NUM_PREDICTORS * sizeof(short) * imagesMeta[i].size, hipMemcpyDeviceToHost));
		cout << "DONE" << endl;
	}
}
