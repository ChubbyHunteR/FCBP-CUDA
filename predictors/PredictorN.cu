#include "hip/hip_runtime.h"
#include "PredictorN.h"
#include "../config.h"

typedef unsigned char byte;
namespace {

	__device__ byte predict(byte *iData, unsigned w, unsigned h) {
		unsigned absolutePosition = threadIdx.x + blockIdx.x * THREADS;
		unsigned x = absolutePosition % w;
		unsigned y = absolutePosition / w - 1;
		if(x < w && y < h){
			return iData[y * w + x];
		}
		return 0;
	}

	__global__ void predict(void *diData, void *dPredicted, unsigned w, unsigned h) {
		unsigned absolutePosition = threadIdx.x + blockIdx.x * THREADS;
		if(absolutePosition >= w*h){
			return;
		}

		byte* iData = (byte*) diData;
		byte* predicted = (byte*) dPredicted;

		predicted[absolutePosition] = predict(iData, w, h);
	}
}

void PredictorN::predict(void *diData, void *dPredicted, unsigned w, unsigned h){
	unsigned size = w * h;
	::predict<<<size/THREADS + 1, THREADS>>>(diData, dPredicted, w, h);
}
