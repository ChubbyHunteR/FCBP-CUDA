#include "hip/hip_runtime.h"
#include "PredictorGW.h"
#include "../config.h"

typedef unsigned char byte;
namespace {

	__device__ byte predict(byte *iData, unsigned w, unsigned h) {
		unsigned absolutePosition = threadIdx.x + blockIdx.x * THREADS;
		unsigned x = absolutePosition % w;
		unsigned y = absolutePosition / w - 2;
		short sum = 0;
		if(x < w && y < h){
			sum -= iData[y * w + x];
		}
		++y;
		if(x < w && y < h){
			sum += 2 * iData[y * w + x];
		}

		if(sum < 0){
			return 0;
		}else if(sum > 255){
			return 255;
		}else{
			return sum;
		}
	}

	__global__ void predict(void *diData, void *dPredicted, unsigned w, unsigned h) {
		unsigned absolutePosition = threadIdx.x + blockIdx.x * THREADS;
		if(absolutePosition >= w*h){
			return;
		}

		byte* iData = (byte*) diData;
		byte* predicted = (byte*) dPredicted;

		predicted[absolutePosition] = predict(iData, w, h);
	}
}

void PredictorGW::predict(void *diData, void *dPredicted, unsigned w, unsigned h){
	unsigned size = w * h;
	::predict<<<size/THREADS + 1, THREADS>>>(diData, dPredicted, w, h);
}
